#include <iostream>
#include <cassert>
#include <cutf/type.hpp>

#define CHECK_TYPE(T, cuda_type) \
	static_assert(cutf::type::get_data_type<T>() == cuda_type, "Type unmatched : " #cuda_type)

int main() {
	CHECK_TYPE(float          , HIP_R_32F);
	CHECK_TYPE(double         , HIP_R_64F);
	CHECK_TYPE(std::uint8_t   , HIP_R_8U);
	CHECK_TYPE(std::int8_t    , HIP_R_8I);
	CHECK_TYPE(std::uint16_t  , HIP_R_16U);
	CHECK_TYPE(std::int16_t   , HIP_R_16I);
	CHECK_TYPE(std::uint32_t  , HIP_R_32U);
	CHECK_TYPE(std::int32_t   , HIP_R_32I);
	CHECK_TYPE(std::uint64_t  , HIP_R_64U);
	CHECK_TYPE(std::int64_t   , HIP_R_64I);
	CHECK_TYPE(unsigned       , HIP_R_32U);
	CHECK_TYPE(int            , HIP_R_32I);
	CHECK_TYPE(hipComplex      , HIP_C_32F);
	CHECK_TYPE(hipDoubleComplex, HIP_C_64F);
}
