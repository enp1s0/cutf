#include "hip/hip_runtime.h"
#include <cutf/cufft.hpp>
#include <cutf/memory.hpp>
#include <cutf/hiprand.hpp>
#include <iostream>
#include <vector>
#include <random>

std::string vec_str(const std::vector<std::size_t>& v) {
  std::string str = "";
  for (std::size_t i = 0; i < v.size(); i++) {
    str += std::to_string(v[i]);
    if (i != v.size() - 1) {
      str += ",";
    }
  }
  return str;
}

template <class T>
struct real_t {
  using type = T;
  const static std::size_t num = 1;
};
template <>
struct real_t<hipComplex> {
  using type = float;
  const static std::size_t num = 2;
};
template <>
struct real_t<hipDoubleComplex> {
  using type = float;
  const static std::size_t num = 2;
};

template <class T>
typename real_t<T>::type real_value(const T v) {return v;}
template <>
typename real_t<hipComplex>::type real_value(const hipComplex v) {return v.x;}
template <>
typename real_t<hipDoubleComplex>::type real_value(const hipDoubleComplex v) {return v.x;}

template <class T>
void rand_init(T* const ptr, const std::size_t len) {
	unsigned long long seed = 10;
	auto cugen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_DEFAULT);
	CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*cugen.get(), seed));
	CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), reinterpret_cast<typename real_t<T>::type*>(ptr), len * real_t<T>::num));
}

template <class IN_T, class OUT_T>
void eval(const std::vector<std::size_t>& dims) {
  std::size_t num_elements = 1;
  for (const auto v : dims) {
    num_elements *= v;
  }

  auto in_uptr  = cutf::memory::get_managed_unique_ptr<IN_T >(num_elements);
  auto out_uptr = cutf::memory::get_managed_unique_ptr<OUT_T>(num_elements);
  auto ref_uptr = cutf::memory::get_managed_unique_ptr<OUT_T>(num_elements);

  rand_init(in_uptr.get(), num_elements);

  hipfftHandle plan;
  const auto type = cutf::cufft::get_type<IN_T, OUT_T>();
  switch (dims.size()) {
    case 1:
      CUTF_HANDLE_ERROR(hipfftPlan1d(&plan, dims[0], type, 1));
    case 2:
      CUTF_HANDLE_ERROR(hipfftPlan2d(&plan, dims[0], dims[1], type));
      break;
    case 3:
      CUTF_HANDLE_ERROR(hipfftPlan3d(&plan, dims[0], dims[1], dims[2], type));
      break;
    default:
      break;
  }

  CUTF_CHECK_ERROR(hipfftXtExec(plan, in_uptr.get(), out_uptr.get(),HIPFFT_FORWARD));
  CUTF_CHECK_ERROR(hipfftXtExec(plan, out_uptr.get(), ref_uptr.get(),HIPFFT_BACKWARD));

  CUTF_CHECK_ERROR(hipDeviceSynchronize());

  double max_error = 0;
  for (std::size_t i = 0; i < num_elements; i++) {
    max_error = std::max(static_cast<double>(real_value(in_uptr.get()[i])) - real_value(ref_uptr.get()[i]), max_error);
  }

  std::printf("dims=(%s), max_error=%e\n", vec_str(dims).c_str(), max_error);
}

int main() {
  for (const auto dims : std::vector<std::vector<std::size_t>>{{1024}, {256, 256}, {32, 32, 32}}) {
    eval<float    , hipComplex>(dims);
    eval<hipComplex, hipComplex>(dims);
    eval<hipComplex, float    >(dims);
    eval<double         , hipDoubleComplex>(dims);
    eval<hipDoubleComplex, hipDoubleComplex>(dims);
    eval<hipDoubleComplex, double         >(dims);
  }
}
