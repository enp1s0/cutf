#include <cutf/memory.hpp>

int main() {
	constexpr unsigned N = 1u << 20;

	// Standar device memory allocation
	auto device_ptr = cutf::memory::malloc<float>(N);
	cutf::memory::free(device_ptr);

	// Managed memory allocation
	auto managed_ptr = cutf::memory::malloc_managed<float>(N);
	cutf::memory::free(managed_ptr);

	// Host memory allocation
	auto host_ptr = cutf::memory::malloc_host<float>(N);
	cutf::memory::free_host(host_ptr);

	// Async device memory allocation
	hipStream_t hip_stream;
	CUTF_CHECK_ERROR(hipStreamCreate(&hip_stream));

	auto async_device_ptr = cutf::memory::malloc_async<float>(N, hip_stream);
	cutf::memory::free_async(async_device_ptr, hip_stream);

	CUTF_CHECK_ERROR(hipStreamDestroy(hip_stream));
}
