#include "hip/hip_runtime.h"
#include <iostream>
#include <cutf/type.hpp>
#include <cutf/debug/type.hpp>

template <class T>
__global__ void print_typename_test() {
	const auto type_string = cutf::debug::type::get_type_name<T>();
	printf("%s\n", type_string);
}

int main() {
	print_typename_test<double><<<1, 1>>>();
	print_typename_test<float ><<<1, 1>>>();
	print_typename_test<half  ><<<1, 1>>>();
	print_typename_test<half2 ><<<1, 1>>>();
	hipDeviceSynchronize();
}
