#include <iostream>
#include <cutf/memory.hpp>
#include <cutf/debug/matrix.hpp>

constexpr unsigned N = 4;

int main() {
	float h_mat[N * N];
	for (unsigned i = 0; i < N * N; i++) {
		h_mat[i] = i;
	}
	std::printf("# print_matrix\n");
	cutf::debug::print::print_matrix(h_mat, N, N);
	cutf::debug::print::print_matrix_hex(h_mat, N, N);
	cutf::debug::print::print_numpy_matrix(h_mat, N, N);
	cutf::debug::print::print_matrix(h_mat, N, N, N);
	cutf::debug::print::print_matrix_hex(h_mat, N, N, N);
	cutf::debug::print::print_numpy_matrix(h_mat, N, N, N);

	auto d_uptr = cutf::memory::get_device_unique_ptr<float>(N * N);
	cutf::memory::copy(d_uptr.get(), h_mat, N * N);

	std::printf("# print_matrix_from_host\n");
	cutf::debug::print::print_matrix_from_host(d_uptr.get(), N, N);
	cutf::debug::print::print_matrix_hex_from_host(d_uptr.get(), N, N);
	cutf::debug::print::print_numpy_matrix_from_host(d_uptr.get(), N, N);
	cutf::debug::print::print_matrix_from_host(d_uptr.get(), N, N, N);
	cutf::debug::print::print_matrix_hex_from_host(d_uptr.get(), N, N, N);
	cutf::debug::print::print_numpy_matrix_from_host(d_uptr.get(), N, N, N);
}
