#include <random>
#include <cutf/cublas.hpp>
#include <cutf/type.hpp>
#include <cutf/memory.hpp>

#define CUBLAS_ERROR_HANDLE(status) cutf::cublas::error::check(status, __FILE__, __LINE__, __func__)

using compute_t = float;
const std::size_t N = 1 << 10;

int main(){
	auto hA = cutf::cuda::memory::get_host_unique_ptr<compute_t>(N * N);
	auto hB = cutf::cuda::memory::get_host_unique_ptr<compute_t>(N * N);
	auto hC = cutf::cuda::memory::get_host_unique_ptr<compute_t>(N * N);
	auto dA = cutf::cuda::memory::get_device_unique_ptr<compute_t>(N * N);
	auto dB = cutf::cuda::memory::get_device_unique_ptr<compute_t>(N * N);
	auto dC = cutf::cuda::memory::get_device_unique_ptr<compute_t>(N * N);

	std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
	std::mt19937 mt(std::random_device{}());

	for(auto i = decltype(N)(0); i < N * N; i++){
		hA.get()[i] = dist(mt);
		hB.get()[i] = dist(mt);
	}

	cutf::cuda::memory::copy(dA.get(), hA.get(), N * N);
	cutf::cuda::memory::copy(dB.get(), hB.get(), N * N);

	auto cublas_handle = cutf::cublas::get_cublas_unique_ptr();
	compute_t alpha = cutf::cuda::type::cast<compute_t>(1.0f);
	compute_t beta = cutf::cuda::type::cast<compute_t>(1.0f);

	CUBLAS_ERROR_HANDLE(cutf::cublas::gemm(
				*cublas_handle.get(),
				HIPBLAS_OP_N, HIPBLAS_OP_N,
				N, N, N,
				&alpha,
				dA.get(), N,
				dB.get(), N,
				&beta,
				dC.get(), N
				));

	cutf::cuda::memory::copy(hC.get(), dC.get(), N * N);
}
