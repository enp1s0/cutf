#include "hip/hip_runtime.h"
#include <iostream>
#include <cutf/memory.hpp>
#include <cutf/math.hpp>
#include <cutf/type.hpp>

constexpr std::size_t N = 1 << 4;
constexpr std::size_t threads_per_block = 1 << 7;

namespace{
template <class T>
__global__ void abs_kernel(T* const m, const std::size_t N){
	auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= N) return;
	m[tid] = cutf::math::abs(*(m + tid));
}

template <class T>
void test_abs(const std::size_t N){
	std::cout<<"# "<<cutf::type::get_type_name<T>()<<" test --"<<std::endl;
	auto dM = cutf::memory::get_device_unique_ptr<T>(N);
	auto hM = cutf::memory::get_host_unique_ptr<T>(N);
	std::cout<<"m = ";
	for(auto i = decltype(N)(0); i < N; i++){
		hM.get()[i] = cutf::type::cast<T>(N/2.0f - i);
		std::cout<<cutf::type::cast<float>(hM.get()[i])<<" ";
	}
	std::cout<<std::endl;
	cutf::memory::copy(dM.get(), hM.get(), N);

	abs_kernel<T><<<(N + threads_per_block - 1)/threads_per_block, threads_per_block>>>(dM.get(), N);

	cutf::memory::copy(hM.get(), dM.get(), N);

	std::cout<<"|m| = ";
	for(auto i = decltype(N)(0); i < N; i++){
		std::cout<<cutf::type::cast<float>(hM.get()[i])<<" ";
	}
	std::cout<<std::endl;
}
}

int main(){
	test_abs<half>(N);
	test_abs<float>(N);
	test_abs<double>(N);
}
