#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <cutf/type.hpp>

#define CHECK_TYPE(T, cuda_type) \
	static_assert(cutf::type::get_data_type<T>() == cuda_type, "Type unmatched : " #cuda_type)

#define TYPE_CAST_TEST(DST, SRC) \
{const SRC a = 1; const auto v = cutf::type::cast<DST>(a);CUTF_UNUSED(v);}

#ifdef __CUTF_FP8_EXIST__
#define TYPE_CAST_TEST_1(DST) \
	TYPE_CAST_TEST(DST, half           ) \
	TYPE_CAST_TEST(DST, __hip_bfloat16  ) \
	TYPE_CAST_TEST(DST, float          ) \
	TYPE_CAST_TEST(DST, double         ) \
	TYPE_CAST_TEST(DST, std::uint8_t   ) \
	TYPE_CAST_TEST(DST, std::int8_t    ) \
	TYPE_CAST_TEST(DST, std::uint16_t  ) \
	TYPE_CAST_TEST(DST, std::int16_t   ) \
	TYPE_CAST_TEST(DST, std::uint32_t  ) \
	TYPE_CAST_TEST(DST, std::int32_t   ) \
	TYPE_CAST_TEST(DST, std::uint64_t  ) \
	TYPE_CAST_TEST(DST, std::int64_t   ) \
	TYPE_CAST_TEST(DST, unsigned       ) \
	TYPE_CAST_TEST(DST, int            ) \
	TYPE_CAST_TEST(DST, __hip_fp8_e5m2_fnuz  ) \
	TYPE_CAST_TEST(DST, __hip_fp8_e4m3_fnuz  )

#define TYPE_CAST_TEST_2 \
	TYPE_CAST_TEST_1(half           ) \
	TYPE_CAST_TEST_1(__hip_bfloat16  ) \
	TYPE_CAST_TEST_1(float          ) \
	TYPE_CAST_TEST_1(double         ) \
	TYPE_CAST_TEST_1(std::uint8_t   ) \
	TYPE_CAST_TEST_1(std::int8_t    ) \
	TYPE_CAST_TEST_1(std::uint16_t  ) \
	TYPE_CAST_TEST_1(std::int16_t   ) \
	TYPE_CAST_TEST_1(std::uint32_t  ) \
	TYPE_CAST_TEST_1(std::int32_t   ) \
	TYPE_CAST_TEST_1(std::uint64_t  ) \
	TYPE_CAST_TEST_1(std::int64_t   ) \
	TYPE_CAST_TEST_1(unsigned       ) \
	TYPE_CAST_TEST_1(int            ) \
	TYPE_CAST_TEST_1(__hip_fp8_e5m2_fnuz  ) \
	TYPE_CAST_TEST_1(__hip_fp8_e4m3_fnuz  )
#else
#define TYPE_CAST_TEST_1(DST) \
	TYPE_CAST_TEST(DST, half           ) \
	TYPE_CAST_TEST(DST, __hip_bfloat16  ) \
	TYPE_CAST_TEST(DST, float          ) \
	TYPE_CAST_TEST(DST, double         ) \
	TYPE_CAST_TEST(DST, std::uint8_t   ) \
	TYPE_CAST_TEST(DST, std::int8_t    ) \
	TYPE_CAST_TEST(DST, std::uint16_t  ) \
	TYPE_CAST_TEST(DST, std::int16_t   ) \
	TYPE_CAST_TEST(DST, std::uint32_t  ) \
	TYPE_CAST_TEST(DST, std::int32_t   ) \
	TYPE_CAST_TEST(DST, std::uint64_t  ) \
	TYPE_CAST_TEST(DST, std::int64_t   ) \
	TYPE_CAST_TEST(DST, unsigned       ) \
	TYPE_CAST_TEST(DST, int            )

#define TYPE_CAST_TEST_2 \
	TYPE_CAST_TEST_1(half           ) \
	TYPE_CAST_TEST_1(__hip_bfloat16  ) \
	TYPE_CAST_TEST_1(float          ) \
	TYPE_CAST_TEST_1(double         ) \
	TYPE_CAST_TEST_1(std::uint8_t   ) \
	TYPE_CAST_TEST_1(std::int8_t    ) \
	TYPE_CAST_TEST_1(std::uint16_t  ) \
	TYPE_CAST_TEST_1(std::int16_t   ) \
	TYPE_CAST_TEST_1(std::uint32_t  ) \
	TYPE_CAST_TEST_1(std::int32_t   ) \
	TYPE_CAST_TEST_1(std::uint64_t  ) \
	TYPE_CAST_TEST_1(std::int64_t   ) \
	TYPE_CAST_TEST_1(unsigned       ) \
	TYPE_CAST_TEST_1(int            )
#endif

int main() {
	CHECK_TYPE(half           , HIP_R_16F);
	CHECK_TYPE(__hip_bfloat16  , HIP_R_16BF);
	CHECK_TYPE(float          , HIP_R_32F);
	CHECK_TYPE(double         , HIP_R_64F);
	CHECK_TYPE(std::uint8_t   , HIP_R_8U);
	CHECK_TYPE(std::int8_t    , HIP_R_8I);
	CHECK_TYPE(std::uint16_t  , HIP_R_16U);
	CHECK_TYPE(std::int16_t   , HIP_R_16I);
	CHECK_TYPE(std::uint32_t  , HIP_R_32U);
	CHECK_TYPE(std::int32_t   , HIP_R_32I);
	CHECK_TYPE(std::uint64_t  , HIP_R_64U);
	CHECK_TYPE(std::int64_t   , HIP_R_64I);
	CHECK_TYPE(unsigned       , HIP_R_32U);
	CHECK_TYPE(int            , HIP_R_32I);
	CHECK_TYPE(hipComplex      , HIP_C_32F);
	CHECK_TYPE(hipDoubleComplex, HIP_C_64F);

#ifdef __CUTF_FP8_EXIST__
	CHECK_TYPE(__hip_fp8_e5m2_fnuz  , HIP_R_8F_E5M2);
	CHECK_TYPE(__hip_fp8_e4m3_fnuz  , HIP_R_8F_E4M3);
#endif

  TYPE_CAST_TEST_2
}
