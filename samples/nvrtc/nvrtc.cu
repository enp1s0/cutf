#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <cutf/type.hpp>
#include <cutf/memory.hpp>
#include <cutf/cuda.hpp>
#include <cutf/nvrtc.hpp>
#include <cutf/error.hpp>

int main(){
	const std::size_t N = 1 << 8;
	const std::string code = R"(
extern "C"
__global__ void kernel(float *a, float *b){
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	a[tid] = b[tid];
}
)";
	CUTF_CHECK_ERROR(hipInit(0));
	auto cu_context = cutf::cu::get_context_unique_ptr();
	cutf::cu::create_context(cu_context.get(), 0);
	auto cu_module = cutf::cu::get_module_unique_ptr();

	const auto ptx_code = cutf::nvrtc::get_ptx(
				"kernel.cu",
				code,
				{"--arch=sm_60"},
				{},
				false
			);

	const auto function = cutf::nvrtc::get_function(
			ptx_code,
			"kernel",
			cu_module.get()
			);

	std::cout<<"/* -- PTX" <<std::endl
			<<ptx_code<<std::endl
			<<" -- */"<<std::endl;

	auto hAB = cutf::memory::get_host_unique_ptr<float>(N);
	for(auto i = decltype(N)(0); i < N; i++) hAB.get()[i] = static_cast<float>(i);
	auto dA = cutf::memory::get_device_unique_ptr<float>(N);
	auto dB = cutf::memory::get_device_unique_ptr<float>(N);
	cutf::memory::copy(dB.get(), hAB.get(), N);

	const float * dA_ptr = dA.get();
	const float * dB_ptr = dB.get();

	std::cout<<"# -- kernel launch" <<std::endl;

	cutf::nvrtc::launch_function(
			function,
			{&dA_ptr, &dB_ptr},
			N,
			1
			);

	cutf::memory::copy(hAB.get(), dA.get(), N);

	std::cout<<"/* -- kernel result" <<std::endl;
	for(auto i = decltype(N)(0); i < N; i++) {
		std::cout<<hAB.get()[i] << " ";
		if(i % 8 == 7)
			std::cout<<std::endl;
	}
	std::cout<<" -- */"<<std::endl;
}
